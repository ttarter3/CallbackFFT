// Project Header
#include "axpy.hpp"

// Project Kernel
#include "axpy.cuh"
#include "gpuErrorChk.h"

// Standard Headers
#include <iostream>

#define THREADSPERBLOCK 1024

// Constructors
template<typename T>
Axpy<T>::Axpy(int N, int deviceID ) : N_(N), device_id_(deviceID) {
  int device_Count;
  hipGetDeviceCount(&device_Count);
  printf("\n\nNumber of GPUs: %d\n", device_Count);
  // Device Selection
  hipDeviceProp_t deviceProp;
  int status = hipSetDevice(device_id_);
  hipGetDevice(&device_id_);   	
  hipGetDeviceProperties(&deviceProp, device_id_);
  printf("Using device %d: %s\n", device_id_, deviceProp.name);



	gpuErrchk( hipMalloc((void**)&d_x_, N_ * sizeof(T)) );
	gpuErrchk( hipMalloc((void**)&d_y_, N_ * sizeof(T)) );
};  // Default constructor

// Destructor
template<typename T>
Axpy<T>::~Axpy() {  
	gpuErrchk( hipFree(d_x_) );
	gpuErrchk( hipFree(d_y_) );
};

template<typename T>
void Axpy<T>::Load(T * h_x, T * h_y) {  
	// Copy input vectors from host to device
	gpuErrchk( hipMemcpy(d_x_, h_x, N_ * sizeof(T), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(d_y_, h_y, N_ * sizeof(T), hipMemcpyHostToDevice) );
};

template<typename T>
void Axpy<T>::Execute(T a) {  
  dim3 threads_per_block(THREADSPERBLOCK);
  dim3 blocks_per_grid((N_ + THREADSPERBLOCK - 1) / THREADSPERBLOCK);
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  hipEventRecord(start);
  saxpy<<<blocks_per_grid, threads_per_block>>>(a, d_x_, d_y_, N_);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  
  printf("Operating Time(millsec): %f\n", milliseconds);
  
	// Wait for kernel to finish
	gpuErrchk( hipGetLastError() );
	gpuErrchk( hipDeviceSynchronize() );
};

template<typename T>
void Axpy<T>::Purge(T * h_y) {  
	// Copy output vector from device to host
	gpuErrchk( hipMemcpy(h_y, d_y_, N_ * sizeof(T), hipMemcpyDeviceToHost) );
};

template class Axpy<float>;
template class Axpy<double>;


